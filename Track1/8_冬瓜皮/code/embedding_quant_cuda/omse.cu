#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <cfloat>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const int ITER = 100;
const float RATE = 1 / float(ITER);


// __global__ void min_max_block(float *weight, float *weight_max, float *weight_min) {
//     int row = blockIdx.x;
//     int tid = threadIdx.x;

//     __shared__ float max_in_block;
//     __shared__ float min_in_block;
//     max_in_block = -FLT_MAX;
//     min_in_block = FLT_MAX;

//     max_in_block = fmaxf(max_in_block, weight[row * blockDim.x + tid]);
//     min_in_block = fminf(min_in_block, weight[row * blockDim.x + tid]);

//     // 每个线程只需要比较自己的最大值和最小值
//     __syncthreads();
//     weight_max[row] = fmaxf(0, max_in_block);
//     weight_min[row] = fminf(0, min_in_block);
// }
__global__ void min_max_block(float *weight, float *weight_max, float *weight_min) {
    int row = blockIdx.x;
    int tid = threadIdx.x;

    extern __shared__ float sharedData[];
    float *smin = sharedData;
    float *smax = sharedData + blockDim.x;

    smin[tid] = weight[row * blockDim.x + tid];
    smax[tid] = weight[row * blockDim.x + tid];

    __syncthreads();

    for (int s=blockDim.x /2; s>0; s>>=1){
        if (tid < s) {
            smin[tid] = fminf(smin[tid], smin[tid + s]);
            smax[tid] = fmaxf(smax[tid], smax[tid + s]);
        }

        //这里就是要求所有线程都执行完一个循环，然后再继续下一个循环
        __syncthreads();
    }

    if (tid == 0) {
        weight_max[row] = fmaxf(0.0, smax[0]);
        weight_min[row] = fminf(0.0, smin[0]);
    }
}

__device__ __forceinline__ float mse_calc(float now_mse, float weight1, float weight2, int count) {
    /*  避免和溢出
        self.H *= self.nsamples / (self.nsamples + tmp)
        self.nsamples += tmp
        self.H += self.nsamples * (inp.matmul(inp.t())).double()

        x = x * n/(n+1)
        n = n + 1
        x += n * y

        mean = (x1 + x2 + ... + xn) / n
        mean = [(x1 + x2) / 2] * 2 + x3] / 3 ...
     */

    float mse_tmp = (weight1 - weight2) * (weight1 - weight2);
    now_mse = now_mse * (count - 1);
    now_mse = (now_mse + mse_tmp) / float(count);

    return now_mse;
}

__device__ __forceinline__ float fake_quant(float weight, float scale, float zp, int q_min, int q_max) {
    float q_weight = roundf(weight / scale + zp);
    q_weight = fminf(q_max, fmaxf(q_min, q_weight));
    float dq_weight = q_weight * scale - zp;
    return dq_weight;
}


__global__ void quant_block_kernel_asym(float *weight, float *scale_arr, float *zp_arr,
                                        float *weight_max, float *weight_min,
                                        int row_count, int col_count,
                                        int q_min, int q_max) {
    /*
     * 代码有很多地方需要优化，要问 chatgpt
     */
    // 每个线程处理一行数据
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (row_id < row_count) {
        int maxq = q_max - q_min;

        float scale = 0.0;
        float zp = 0.0;

        float best_mse = FLT_MAX;
        float best_scale = 0;
        float best_zp = 0;

        for (int i = 0; i < ITER; i++) {
            float now_mse = 0;
            float max_val = weight_max[row_id];
            float min_val = weight_min[row_id];
            float percent = 1.0 - i * RATE;
            max_val = max_val * percent;
            min_val = min_val * percent;
            scale = (max_val - min_val) / (float) maxq;
            zp = -min_val / scale;

            for (int j = 0; j < col_count; j++) {
                float now_weight = weight[row_id * col_count + j];
                float q_weight = fake_quant(now_weight, scale, zp, q_min, q_max);
                now_mse = mse_calc(now_mse, now_weight, q_weight, j + 1);
            }
            if (now_mse < best_mse) {
                best_mse = now_mse;
                best_scale = scale;
                best_zp = zp;
            }
        }
        scale_arr[row_id] = best_scale;
        zp_arr[row_id] = best_zp;
    }
}


__global__ void get_best_percentile_asym(float *weight, float *percent_attr,
                                        float *weight_max, float *weight_min,
                                        int row_count, int col_count,
                                        int q_min, int q_max) {
    /*
     * 代码有很多地方需要优化，要问 chatgpt
     */
    // 每个线程处理一行数据
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (row_id < row_count) {
        int maxq = q_max - q_min;

        float scale = 0.0;
        float zp = 0.0;

        float best_mse = FLT_MAX;
        float best_percent = 0.0;

        for (int i = 0; i < ITER; i++) {
            float now_mse = 0;
            float max_val = weight_max[row_id];
            float min_val = weight_min[row_id];
            float percent = 1.0 - i * RATE;
            max_val = max_val * percent;
            min_val = min_val * percent;
            scale = (max_val - min_val) / (float) maxq;
            zp = -min_val / scale;

            for (int j = 0; j < col_count; j++) {
                float now_weight = weight[row_id * col_count + j];
                float q_weight = fake_quant(now_weight, scale, zp, q_min, q_max);
                now_mse = mse_calc(now_mse, now_weight, q_weight, j + 1);
            }
            if (now_mse < best_mse) {
                best_mse = now_mse;
                best_percent = percent;
            }
        }
        percent_attr[row_id] = best_percent;
    }
}
__global__ void get_best_percentile_sym(float *weight, float *percent_attr,
                                        float *weight_max, float *weight_min,
                                        int row_count, int col_count,
                                        int q_min, int q_max) {
    /*
     * 代码有很多地方需要优化，要问 chatgpt
     */
    // 每个线程处理一行数据
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (row_id < row_count) {
        int maxq = q_max - q_min;

        float scale = 0.0;
        float zp = 0.0;

        float best_mse = FLT_MAX;
        float best_percent = 0.0;

        for (int i = 0; i < ITER; i++) {
            float now_mse = 0;
            float max_val = weight_max[row_id];
            float min_val = weight_min[row_id];
            float percent = 1.0 - i * RATE;
            max_val = max_val * percent;
            min_val = min_val * percent;

            float f_max = fmaxf(fabs(max_val), fabs(min_val));
            scale = 2 * f_max / (float) maxq;
            zp = 0;

            for (int j = 0; j < col_count; j++) {
                float now_weight = weight[row_id * col_count + j];
                float q_weight = fake_quant(now_weight, scale, zp, q_min, q_max);
                now_mse = mse_calc(now_mse, now_weight, q_weight, j + 1);
            }
            if (now_mse < best_mse) {
                best_mse = now_mse;
                best_percent = percent;
            }
        }
        percent_attr[row_id] = best_percent;
    }
}

__global__ void quant_block_kernel_sym(float *weight, float *scale_arr, float *zp_arr,
                                       float *weight_max, float *weight_min,
                                       int row_count, int col_count,
                                       int q_min, int q_max) {
    /*
     * 代码有很多地方需要优化，要问 chatgpt
     */
    // 每个线程处理一行数据
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (row_id < row_count) {
        int maxq = q_max - q_min;
        float scale = 0.0;
        float zp = 0;

        float best_mse = FLT_MAX;
        float best_scale = 0;
        float best_zp = 0;

        for (int i = 0; i < ITER; i++) {
            float now_mse = 0;
            float max_val = weight_max[row_id];
            float min_val = weight_min[row_id];
            float percent = 1.0 - i * RATE;
            max_val = max_val * percent;
            min_val = min_val * percent;

            float f_max = fmaxf(fabs(max_val), fabs(min_val));
            scale = 2 * f_max / (float) maxq;
            zp = 0;

            for (int j = 0; j < col_count; j++) {
                float now_weight = weight[row_id * col_count + j];
                float q_weight = fake_quant(now_weight, scale, zp, q_min, q_max);
                now_mse = mse_calc(now_mse, now_weight, q_weight, j + 1);
            }
            if (now_mse < best_mse) {
                best_mse = now_mse;
                best_scale = scale;
                best_zp = zp;
            }
        }
        scale_arr[row_id] = best_scale;
        zp_arr[row_id] = best_zp;
    }
}

void quant_block(torch::Tensor weight, torch::Tensor scale, torch::Tensor zp, int bits = 8, bool sym = true) {
    /* 整体流程分为：
     * 1. 计算每个 block （行）的最大值和最小值
     * 2. 根据最大值和最小值，进行 omse 搜索
     *
     * 输入：
     *  weight: 二维张量，形状为 [block_count, block_size]
     * 输出：
     *  scale: 一维张量，形状为 [block_count]
     *  zp: 一维张量，形状为 [block_count]
     */
    if (!weight.is_cuda()) {
        throw std::invalid_argument("weight must be in cuda");
    }
    if (!scale.is_cuda()) {
        throw std::invalid_argument("scale must be in cuda");
    }
    if (!zp.is_cuda()) {
        throw std::invalid_argument("zp must be in cuda");
    }

    if (weight.dim() != 2) {
        throw std::invalid_argument("weight must be 2D tensor");
    }
    int block_count = weight.size(0);
    int block_size = weight.size(1);
    // 确保 scale 和 zp 的形状和 weight 的形状一致
    if (scale.dim() != 1 || scale.size(0) != block_count) {
        throw std::invalid_argument("scale must be 1D tensor with the same size as the first dimension of weight");
    }
    if (zp.dim() != 1 || zp.size(0) != block_count) {
        throw std::invalid_argument("zp must be 1D tensor with the same size as the first dimension of weight");
    }

    if (bits < 2 || bits > 8) {
        throw std::invalid_argument("bits must be between 2 and 8");
    }

    float *weight_data = weight.data_ptr<float>();
    float *scale_data = scale.data_ptr<float>();
    float *zp_data = zp.data_ptr<float>();

    int q_min, q_max;
    if (sym) {
        q_min = -(1 << (bits - 1)) + 1;
        q_max = (1 << (bits - 1)) - 1;
    } else {
        q_min = 0;
        q_max = (1 << bits) - 1;
    }

    /*
     * 每个线程块负责处理一行数据，总共需要 block_count 个线程块, 也就是 grid 的大小
     * 每个线程块中的线程数为 block_size，也就是 block 的大小
     */
    torch::Tensor weight_max = torch::zeros({block_count}, torch::dtype(torch::kFloat32).device(weight.device()));
    torch::Tensor weight_min = torch::zeros({block_count}, torch::dtype(torch::kFloat32).device(weight.device()));


    // 这里乘以 2 是因为要存储最大值和最小值
    // 注意，由于我们让每个线程块处理一行，并且每个线程处理一个元素，所以情况得到简化
    // 因为不需要考虑矩阵列数比线程数多的情况
    int sharedMemSize = 2 * block_size * sizeof(float);
    // min_max_block<<<block_count, block_size, sharedMemSize>>>(weight_data,
    //     weight_max.data_ptr<float>(),
    //     weight_min.data_ptr<float>());
    min_max_block<<<block_count, block_size, sharedMemSize>>>(weight_data,
                                               weight_max.data_ptr<float>(),
                                               weight_min.data_ptr<float>());


    /* 为了最大化性能，块内线程数要大于 32, 可以取 64 或 128等
     * 量化以 block 为单位进行，如果此时还是每个 block 处理一行，那么每个 block 的线程数只有 1，这样会导致性能下降
     */
    int kernel_block_size = 256;
    int kernel_grid_size = (block_count + kernel_block_size - 1) / kernel_block_size;


    // 矩阵总共多少行
    int rowCount = block_count;
    int colCount = block_size;
    if (sym) {
        quant_block_kernel_sym<<<kernel_grid_size, kernel_block_size>>>(weight_data,
                                                                        scale_data, zp_data,
                                                                        weight_max.data_ptr<float>(),
                                                                        weight_min.data_ptr<float>(),
                                                                        rowCount, colCount,
                                                                        q_min, q_max);
    } else {
        quant_block_kernel_asym<<<kernel_grid_size, kernel_block_size>>>(weight_data,
                                                                         scale_data, zp_data,
                                                                         weight_max.data_ptr<float>(),
                                                                         weight_min.data_ptr<float>(),
                                                                         rowCount, colCount,
                                                                         q_min, q_max);
    }
}


void best_percentile(torch::Tensor weight, torch::Tensor percentile_attr, int bits = 8, bool sym = true) {
    /* 整体流程分为：
     * 1. 计算每个 block （行）的最大值和最小值
     * 2. 根据最大值和最小值，进行 omse 搜索
     *
     * 输入：
     *  weight: 二维张量，形状为 [block_count, block_size]
     * 输出：
     *  scale: 一维张量，形状为 [block_count]
     *  zp: 一维张量，形状为 [block_count]
     */
    if (!weight.is_cuda()) {
        throw std::invalid_argument("weight must be in cuda");
    }
    if (!percentile_attr.is_cuda()) {
        throw std::invalid_argument("scale must be in cuda");
    }

    if (weight.dim() != 2) {
        throw std::invalid_argument("weight must be 2D tensor");
    }
    int block_count = weight.size(0);
    int block_size = weight.size(1);
    // 确保 scale 和 zp 的形状和 weight 的形状一致
    if (percentile_attr.dim() != 1 || percentile_attr.size(0) != block_count) {
        throw std::invalid_argument("scale must be 1D tensor with the same size as the first dimension of weight");
    }

    if (bits < 2 || bits > 8) {
        throw std::invalid_argument("bits must be between 2 and 8");
    }


    float *weight_data = weight.data_ptr<float>();
    float *percentile_data = percentile_attr.data_ptr<float>();
    int q_min, q_max;
    if (sym) {
        q_min = -(1 << (bits - 1)) + 1;
        q_max = (1 << (bits - 1)) - 1;
    } else {
        q_min = 0;
        q_max = (1 << bits) - 1;
    }

    /*
     * 每个线程块负责处理一行数据，总共需要 block_count 个线程块, 也就是 grid 的大小
     * 每个线程块中的线程数为 block_size，也就是 block 的大小
     */
    torch::Tensor weight_max = torch::zeros({block_count}, torch::dtype(torch::kFloat32).device(weight.device()));
    torch::Tensor weight_min = torch::zeros({block_count}, torch::dtype(torch::kFloat32).device(weight.device()));


    // 这里乘以 2 是因为要存储最大值和最小值
    // 注意，由于我们让每个线程块处理一行，并且每个线程处理一个元素，所以情况得到简化
    // 因为不需要考虑矩阵列数比线程数多的情况
    int sharedMemSize = 2 * block_size * sizeof(float);
    // min_max_block<<<block_count, block_size, sharedMemSize>>>(weight_data,
    //     weight_max.data_ptr<float>(),
    //     weight_min.data_ptr<float>());
    min_max_block<<<block_count, block_size, sharedMemSize>>>(weight_data,
                                               weight_max.data_ptr<float>(),
                                               weight_min.data_ptr<float>());


    /* 为了最大化性能，块内线程数要大于 32, 可以取 64 或 128等
     * 量化以 block 为单位进行，如果此时还是每个 block 处理一行，那么每个 block 的线程数只有 1，这样会导致性能下降
     */
    int kernel_block_size = 256;
    int kernel_grid_size = (block_count + kernel_block_size - 1) / kernel_block_size;

    // 矩阵总共多少行
    int rowCount = block_count;
    int colCount = block_size;
    if (sym) {
        get_best_percentile_sym<<<kernel_grid_size, kernel_block_size>>>(weight_data,
                                                                        percentile_data,
                                                                        weight_max.data_ptr<float>(),
                                                                        weight_min.data_ptr<float>(),
                                                                        rowCount, colCount,
                                                                        q_min, q_max);
    } else {
        get_best_percentile_asym<<<kernel_grid_size, kernel_block_size>>>(weight_data,
                                                                         percentile_data, 
                                                                         weight_max.data_ptr<float>(),
                                                                         weight_min.data_ptr<float>(),
                                                                         rowCount, colCount,
                                                                         q_min, q_max);
    }
}
